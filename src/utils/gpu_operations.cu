#include "../../include/gpu_operations.cuh"
#include <ctime>

namespace RuNet {
namespace Utils {
void setGpuValue(float *x, int n, float val) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    x[i] = val;
  }
}

void setGpuNormalValue(float *x, int n, float mean, float stddev) {
  hiprandGenerator_t rand_gen;
  hiprandCreateGenerator(&rand_gen, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(rand_gen, time(0));
  hiprandGenerateNormal(rand_gen, x, n, mean, stddev);
  hiprandDestroyGenerator(rand_gen);
}
};  // namespace Utils
};  // namespace RuNet