#include "hip/hip_runtime.h"
#include "utils/gpu_operations.cuh"

namespace RuNet {
namespace Utils {

__global__ void setGpuValueHelper(float *x, int n, float val) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    x[i] = val;
  }
}

void setGpuValue(float *x, int n, int batch_size, float val) {
  setGpuValueHelper<<<std::ceil((1.0f * batch_size) / (1.0f * Constants::CudaBandWidth)), Constants::CudaBandWidth>>>(x, n, val);
}

void setGpuNormalValue(float *x, int n, float mean, float stddev) {
  hiprandGenerator_t rand_gen;
  hiprandCreateGenerator(&rand_gen, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(rand_gen, time(0));
  hiprandGenerateNormal(rand_gen, x, n, mean, stddev);
  hiprandDestroyGenerator(rand_gen);
}
};  // namespace Utils
};  // namespace RuNet