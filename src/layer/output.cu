#include "hip/hip_runtime.h"
#include <runet/layer/softmax.cuh>
#include <runet/layer/output.cuh>

namespace RuNet {
  template<>
  void Output<Softmax>::forward(const Tensor &tensor) {
    output_layer.forward(tensor);
  }

  template<>
  void Output<Softmax>::backward(Tensor &) {
    softmaxBackward<<<std::ceil((1.0f * output_layer.getBatchSize()) / (1.0f * Constants::CudaBandWidth)), Constants::CudaBandWidth>>>(label_p->data(), label_p->size(),
                                                                                                                                       output_layer.getBatchSize(), diff_for_prev.data());
  }
}